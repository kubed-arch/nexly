#include "hip/hip_runtime.h"
#include "../common/cudaUtils.h"
#include "../common/reduceKernelUtils.cuh"
#include "stopCriteriaKernels.h"

using namespace nexly::common;

namespace nexly
{
namespace kernels
{
__global__ void stopWordsCriterion(int32_t const** outputIds, int32_t const** parentIds, int32_t const** stopWords,
    FinishedState* finished, int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t const* stopWordsLens,
    int32_t batchSize, int32_t beamWidth, int32_t maxSeqLen)
{
    int32_t const id = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t const batchIdx = blockIdx.y / beamWidth;
    int32_t const beamIdx = blockIdx.y % beamWidth;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
    auto const batchBeamIdx = batchSlot * beamWidth + beamIdx;

    auto const* baseStopWords = stopWords[batchSlot];
    auto const stopWordsLen = stopWordsLens[batchSlot];
    auto const* baseOffsets = baseStopWords + stopWordsLen;

    if (id >= stopWordsLen || baseOffsets[id] < 0)
    {
        return;
    }

    auto const itemEnd = baseOffsets[id];
    auto const itemStart = (id > 0) ? baseOffsets[id - 1] : 0;
    auto const itemSize = itemEnd - itemStart;

    bool shouldStop = false;

    auto const currentStep = sequenceLengths[batchBeamIdx] - 1;
    if (currentStep + 1 >= itemSize)
    {
        shouldStop = true;
        auto parentId = beamIdx;
        bool const gatherBeam = beamWidth > 1;

        for (int32_t tokenIdx = itemSize - 1; tokenIdx >= 0; tokenIdx--)
        {
            auto const previousToken
                = outputIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];
            if (previousToken != baseStopWords[itemStart + tokenIdx])
            {
                shouldStop = false;
                break;
            }
            if (gatherBeam)
            {
                parentId = parentIds == nullptr
                    ? 0
                    : parentIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];

                if (parentId < 0 || parentId >= beamWidth)
                {
                    shouldStop = false;
                    break;
                }
            }
        }
    }

    if (shouldStop)
    {
        finished[batchSlot * beamWidth + beamIdx].setFinishedStopWords();
    }
}

void invokeStopWordsCriterion(int32_t const** outputIds, int32_t const** parentIds, int32_t const** stopWords,
    FinishedState* finished, int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t const* stopWordsLen,
    int32_t maxStopWordsLen, int32_t batchSize, int32_t beamWidth, int32_t maxSeqLen, hipStream_t stream)
{
    dim3 block, grid;
    constexpr int32_t maxBlockSize{256};

    block.x = min(((maxStopWordsLen + 32 - 1) / 32) * 32, maxBlockSize);
    grid.x = (maxStopWordsLen + block.x - 1) / block.x;
    grid.y = batchSize * beamWidth;

    stopWordsCriterion<<<grid, block, 0, stream>>>(outputIds, parentIds, stopWords, finished, sequenceLengths,
        batchSlots, stopWordsLen, batchSize, beamWidth, maxSeqLen);
    sync_check_cuda_error();
}

__global__ void lengthCriterion(FinishedState* finished, int32_t* finishedSum, uint32_t const* sequenceLimitLength,
    int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t batchSize, int32_t beamWidth)
{
    int32_t threadFinishedCount = 0;
    auto const batchIdx = blockIdx.x;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;

    for (int32_t beamIdx = threadIdx.x; beamIdx < beamWidth; beamIdx += blockDim.x)
    {
        auto const batchSlotBeamWidthIdx = batchSlot * beamWidth + beamIdx;

        auto finishState = finished[batchSlotBeamWidthIdx];

        if (sequenceLengths[batchSlotBeamWidthIdx] >= sequenceLimitLength[batchSlot])
        {
            finishState.setFinishedMaxLength();
        }
        threadFinishedCount += finishState.isFinished() ? 1 : 0;
        finished[batchSlotBeamWidthIdx] = finishState;
    }

    if (finishedSum)
    {
        int blockFinishedCount = 0;
        if (blockDim.x <= 32)
        {
            blockFinishedCount = warpReduceSum(threadFinishedCount);
        }
        else
        {
            blockFinishedCount = blockReduceSum(threadFinishedCount);
        }
        __syncthreads();

        if (threadIdx.x == 0)
        {
            finishedSum[batchSlot] = blockFinishedCount;
        }
    }
}

void invokeLengthCriterion(FinishedState* finished, int32_t* finishedSum, uint32_t const* sequenceLimitLength,
    int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t batchSize, int32_t beamWidth,
    hipStream_t stream)
{
    dim3 block{min(512, uint32_t(beamWidth))};
    dim3 grid{uint32_t(batchSize)};

    lengthCriterion<<<grid, block, 0, stream>>>(
        finished, finishedSum, sequenceLimitLength, sequenceLengths, batchSlots, batchSize, beamWidth);
    sync_check_cuda_error();
}

}
}
