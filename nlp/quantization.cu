#include "hip/hip_runtime.h"
#include "../common/assert.h"
#include "../common/cudaTypeUtils.cuh"
#include "../common/cudaUtils.h"
#include "../common/reduceKernelUtils.cuh"
#include "quantization.h"

using namespace nexly::common;

namespace nexly
{
    namespace kernels
    {

        __global__ void quantizedKernel(char4* dst, float4 const* src, const int64_t sizeDiv4, float const* scalePtr)
        {
            for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
            {
                float const scale = __ldg(scalePtr);
                char4 tmp;
                const float4 floatTmp = __ldg(src + idx);
                tmp.x = cuda_cast<int8_t>(floatTmp.x * scale);
                tmp.y = cuda_cast<int8_t>(floatTmp.y * scale);
                tmp.z = cuda_cast<int8_t>(floatTmp.z * scale);
                tmp.w = cuda_cast<int8_t>(floatTmp.w * scale);
                dst[idx] = tmp;
            }
        }

        __global__ void quantizedKernel(char4* dst, half2 const* src, const int64_t sizeDiv4, float const* scalePtr)
        {
            for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
            {
                float const scale = __ldg(scalePtr);
                char4 tmp;
                int srcId = idx << 1;

                const uint2 h2 = __ldg(reinterpret_cast<uint2 const*>(src + srcId));

                const half2 half2Tmp = reinterpret_cast<half2 const&>(h2.x);
                const half2 half2Tmp2 = reinterpret_cast<half2 const&>(h2.y);

                tmp.x = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
                tmp.y = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
                tmp.z = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
                tmp.w = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
                dst[idx] = tmp;
            }
        }

        template <typename T>
        void invokeQuantization(
            int8_t* dst, T const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize)
        {
            CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

            int numBlocks{ static_cast<int>((size + 255) / 256) };
            dim3 grid(std::min(numBlocks, maxGridSize));
            CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
            dim3 block(64);
            if (std::is_same_v<T, float>)
            {
                quantizedKernel << <grid, block, 0, stream >> > ((char4*)dst, (float4 const*)src, size / 4, scalePtr);
            }
            else if (std::is_same_v<T, half>)
            {
                quantizedKernel << <grid, block, 0, stream >> > ((char4*)dst, (half2 const*)src, size / 4, scalePtr);
            }
        }

        template void invokeQuantization<float>(
            int8_t* dst, float const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize);

        template void invokeQuantization<half>(
            int8_t* dst, half const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize);

        template <typename T>
        __global__ void perTokenQuantization(
            int8_t* dst, T const* src, const int64_t numRows, const int64_t numCols, float* scalePtr)
        {
            T const* srcRow = src + blockIdx.x * numCols;
            int8_t* dstRow = dst + blockIdx.x * numCols;

            T localMax = 1e-6f;
            for (int i = threadIdx.x; i < numCols; i += blockDim.x)
            {
                localMax = cuda_max(localMax, cuda_abs(srcRow[i]));
            }
            float const rowMax = blockAllReduceMax(cuda_cast<float>(localMax));

            if (threadIdx.x == 0)
            {
                scalePtr[blockIdx.x] = rowMax / 127.f;
            }

            float const scaleOrigQuant = 127.f / rowMax;
            for (int i = threadIdx.x; i < numCols; i += blockDim.x)
            {
                dstRow[i] = cuda_cast<int8_t>(cuda_cast<float>(srcRow[i]) * scaleOrigQuant);
            }
        }

        template <typename T>
        void invokePerTokenQuantization(
            int8_t* dst, T const* src, const int64_t numRows, const int64_t numCols, float* scalePtr, hipStream_t stream)
        {
            const dim3 block(512);
            const dim3 grid(numRows);

            perTokenQuantization << <grid, block, 0, stream >> > (dst, src, numRows, numCols, scalePtr);
        }

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T)                                                                   \
    template void invokePerTokenQuantization(                                                                          \
        int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, hipStream_t stream)

        INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float);
        INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half);
#ifdef ENABLE_BF16
        INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16);
#endif

    }
}