#include "hip/hip_runtime.h"
#include "../common/cudaUtils.h"
#include "../common/reduceKernelUtils.cuh"
#include "stopCriteriaKernels.h"

using namespace nexly::common;

namespace nexly
{
    namespace kernels
    {
        __global__ void stopWordsCriterion(int32_t const** outputIds, int32_t const** parentIds, int32_t const** stopWords,
            FinishedState* finished, int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t const* stopWordsLens,
            int32_t batchSize, int32_t beamWidth, int32_t maxSeqLen)
        {
            int32_t const id = blockIdx.x * blockDim.x + threadIdx.x;
            int32_t const batchIdx = blockIdx.y / beamWidth;
            int32_t const beamIdx = blockIdx.y % beamWidth;
            auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
            auto const batchBeamIdx = batchSlot * beamWidth + beamIdx;

            auto const* baseStopWords = stopWords[batchSlot];
            auto const stopWordsLen = stopWordsLens[batchSlot];
            auto const* baseOffsets = baseStopWords + stopWordsLen;

            if (id >= stopWordsLen || baseOffsets[id] < 0)
            {
                return;
            }

            auto const itemEnd = baseOffsets[id];
            auto const itemStart = (id > 0) ? baseOffsets[id - 1] : 0;
            auto const itemSize = itemEnd - itemStart;

            bool shouldStop = false;

            auto const currentStep = sequenceLengths[batchBeamIdx] - 1;
            if (currentStep + 1 >= itemSize)
            {
                shouldStop = true;
                auto parentId = beamIdx;
                bool const gatherBeam = beamWidth > 1;

                for (int32_t tokenIdx = itemSize - 1; tokenIdx >= 0; tokenIdx--)
                {
                    auto const previousToken
                        = outputIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];
                    if (previousToken != baseStopWords[itemStart + tokenIdx])
                    {
                        shouldStop = false;
                        break;
                    }
                    if (gatherBeam)
                    {
                        parentId = parentIds == nullptr
                            ? 0
                            : parentIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];

                        if (parentId < 0 || parentId >= beamWidth)
                        {
                            shouldStop = false;
                            break;
                        }
                    }
                }
            }

            if (shouldStop)
            {
                finished[batchSlot * beamWidth + beamIdx].setFinishedStopWords();
            }
        }

        void invokeStopWordsCriterion(int32_t const** outputIds, int32_t const** parentIds, int32_t const** stopWords,
            FinishedState* finished, int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t const* stopWordsLen,
            int32_t maxStopWordsLen, int32_t batchSize, int32_t beamWidth, int32_t maxSeqLen, hipStream_t stream)
        {
            constexpr int32_t maxBlockSize{ 256 };
            dim3 block, grid;

            int32_t stopWordsBlocks = (maxStopWordsLen + 32 - 1) / 32;
            block.x = min(stopWordsBlocks * 32, maxBlockSize);
            grid.x = stopWordsBlocks;
            grid.y = batchSize * beamWidth;

            void* args[] = { &outputIds, &parentIds, &stopWords, &finished, &sequenceLengths,
                            &batchSlots, &stopWordsLen, &batchSize, &beamWidth, &maxSeqLen };

            hipError_t launchError = hipLaunchKernel(reinterpret_cast<const void*>(stopWordsCriterion), grid, block, args, 0, stream);
            if (launchError != hipSuccess) {
                std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(launchError) << std::endl;
            }

            hipError_t syncError = hipStreamSynchronize(stream);
            if (syncError != hipSuccess) {
                std::cerr << "CUDA stream synchronization failed: " << hipGetErrorString(syncError) << std::endl;
            }
        }

        __global__ void lengthCriterion(FinishedState* finished, int32_t* finishedSum, uint32_t const* sequenceLimitLength,
            int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t batchSize, int32_t beamWidth)
        {
            int32_t threadFinishedCount = 0;
            auto const batchIdx = blockIdx.x;
            auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;

            for (int32_t beamIdx = threadIdx.x; beamIdx < beamWidth; beamIdx += blockDim.x)
            {
                auto const batchSlotBeamWidthIdx = batchSlot * beamWidth + beamIdx;

                auto finishState = finished[batchSlotBeamWidthIdx];

                if (sequenceLengths[batchSlotBeamWidthIdx] >= sequenceLimitLength[batchSlot])
                {
                    finishState.setFinishedMaxLength();
                }
                threadFinishedCount += finishState.isFinished() ? 1 : 0;
                finished[batchSlotBeamWidthIdx] = finishState;
            }

            if (finishedSum)
            {
                int blockFinishedCount = 0;
                if (blockDim.x <= 32)
                {
                    blockFinishedCount = warpReduceSum(threadFinishedCount);
                }
                else
                {
                    blockFinishedCount = blockReduceSum(threadFinishedCount);
                }
                __syncthreads();

                if (threadIdx.x == 0)
                {
                    finishedSum[batchSlot] = blockFinishedCount;
                }
            }
        }

        void invokeLengthCriterion(FinishedState* finished, int32_t* finishedSum, uint32_t const* sequenceLimitLength,
            int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t batchSize, int32_t beamWidth,
            hipStream_t stream)
        {
            dim3 block(min(512, static_cast<uint32_t>(beamWidth)));
            dim3 grid(static_cast<uint32_t>(batchSize));

            void* args[] = { &finished, &finishedSum, &sequenceLimitLength, &sequenceLengths, &batchSlots,
                            &batchSize, &beamWidth };

            hipError_t launchError = hipLaunchKernel(reinterpret_cast<const void*>(lengthCriterion), grid, block, args, 0, stream);
            if (launchError != hipSuccess) {
                std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(launchError) << std::endl;
            }

            hipError_t syncError = hipStreamSynchronize(stream);
            if (syncError != hipSuccess) {
                std::cerr << "CUDA stream synchronization failed: " << hipGetErrorString(syncError) << std::endl;
            }
        }

    }
}
