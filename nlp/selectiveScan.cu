#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

#ifdef ENABLE_FP8
#include <hip/hip_fp8.h>
#endif

#include "selectiveScan.h"

namespace nexly
{
    namespace kernels
    {

        __device__ float toFloat(float f)
        {
            return f;
        }

        __device__ float toFloat(__half h)
        {
            return __half2float(h);
        }
#ifdef ENABLE_BF16
        __device__ float toFloat(__hip_bfloat16 val)
        {
            return __bfloat162float(val);
        }
#endif

        __device__ void convertAndStore(float* output, float input)
        {
            *output = input;
        }

        __device__ void convertAndStore(__half* output, float input)
        {
            *output = __float2half(input);
        }
#ifdef ENABLE_BF16
        __device__ void convertAndStore(__hip_bfloat16* output, float input)
        {
            *output = __float2bfloat16(input);
        }
#endif

        template <typename input_t, typename weight_t, int DSTATE = 16, int CHANNELS_PER_BLOCK = 128, int STAGES = 12,
            int SEQ_UNROLL = 6>
        __launch_bounds__(256, 1) __global__ void selective_scan_loop_kernel(SSMParamsBase params)
        {
            input_t* output = reinterpret_cast<input_t*>(params.out_ptr);
            input_t* state = reinterpret_cast<input_t*>(params.x_ptr);
            input_t* x = reinterpret_cast<input_t*>(params.u_ptr);
            input_t* dt = reinterpret_cast<input_t*>(params.delta_ptr);
            weight_t* A = reinterpret_cast<weight_t*>(params.A_ptr);
            input_t* B = reinterpret_cast<input_t*>(params.B_ptr);
            input_t* C = reinterpret_cast<input_t*>(params.C_ptr);
            weight_t* D = reinterpret_cast<weight_t*>(params.D_ptr);
            input_t* z = reinterpret_cast<input_t*>(params.z_ptr);
            weight_t* dt_bias = reinterpret_cast<weight_t*>(params.delta_bias_ptr);
            bool dt_softplus = params.delta_softplus;
            int num_tokens = params.seqlen;
            int num_channels = params.dim;

            __shared__ cuda::pipeline_shared_state<cuda::thread_scope::thread_scope_block, STAGES / SEQ_UNROLL> pipeline_state;
            auto block = cooperative_groups::this_thread_block();

            __shared__ __align__(16) input_t sh_B[STAGES][DSTATE];
            __shared__ __align__(16) input_t sh_C[STAGES][DSTATE];

            __shared__ __align__(128) input_t sh_dt[STAGES][CHANNELS_PER_BLOCK];
            __shared__ input_t sh_x[STAGES][CHANNELS_PER_BLOCK];
            __shared__ input_t sh_z[STAGES][CHANNELS_PER_BLOCK];

            __shared__ weight_t sh_D[CHANNELS_PER_BLOCK];
            __shared__ weight_t sh_dt_bias[CHANNELS_PER_BLOCK];

            int const channel = blockIdx.x * blockDim.x + threadIdx.x;
            int const sample = blockIdx.y; // batch id

            int const seq_loops = (num_tokens + SEQ_UNROLL - 1) / SEQ_UNROLL;

            int const input_matrix_row_id = sample * num_tokens;

            if (threadIdx.y == 1)
            {
                sh_dt_bias[threadIdx.x] = dt_bias[channel];
                if (D)
                    sh_D[threadIdx.x] = D[channel];

                cuda::pipeline pipeline = cuda::make_pipeline(block, &pipeline_state, cuda::pipeline_role::producer);

                int stage = 0;
                for (int si = 0; si < seq_loops; si++)
                {

                    pipeline.producer_acquire();

#pragma unroll
                    for (int token_id = si * SEQ_UNROLL; token_id < num_tokens && token_id < (si + 1) * SEQ_UNROLL; token_id++)
                    {

                        input_t* my_B = &B[input_matrix_row_id * DSTATE + token_id * DSTATE];
                        input_t* my_C = &C[input_matrix_row_id * DSTATE + token_id * DSTATE];

                        int block_channel_per_token = blockIdx.x * blockDim.x;
                        int block_channel
                            = input_matrix_row_id * num_channels + token_id * num_channels + block_channel_per_token;

                        if (threadIdx.x < DSTATE)
                            cuda::memcpy_async(&sh_B[stage][threadIdx.x], &my_B[threadIdx.x], sizeof(input_t), pipeline);
                        else if (threadIdx.x >= 32 && threadIdx.x < 32 + DSTATE)
                            cuda::memcpy_async(
                                &sh_C[stage][threadIdx.x - 32], &my_C[threadIdx.x - 32], sizeof(input_t), pipeline);
                        if (sizeof(input_t) == 4)
                        {
                            cuda::memcpy_async(&sh_dt[stage][threadIdx.x],
                                &dt[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                                pipeline);
                            cuda::memcpy_async(&sh_x[stage][threadIdx.x],
                                &x[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                                pipeline);
                            if (z)
                                cuda::memcpy_async(&sh_z[stage][threadIdx.x],
                                    &z[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                                    pipeline);
                        }
                        else
                        {
                            if (threadIdx.x < 32)
                            {
                                int tid = threadIdx.x;
                                float2* block_dt = (float2*)&dt[block_channel];
                                cuda::memcpy_async((float2*)&sh_dt[stage][tid * 4], &block_dt[tid], sizeof(float2), pipeline);
                            }
                            else if (threadIdx.x < 64)
                            {
                                int tid = threadIdx.x - 32;
                                float2* block_x = (float2*)&x[block_channel];
                                cuda::memcpy_async((float2*)&sh_x[stage][tid * 4], &block_x[tid], sizeof(float2), pipeline);
                            }
                            else if (threadIdx.x < 96)
                            {
                                int tid = threadIdx.x - 64;
                                if (z)
                                {
                                    float2* block_z = (float2*)&z[block_channel];
                                    cuda::memcpy_async(
                                        (float2*)&sh_z[stage][tid * 4], &block_z[tid], sizeof(float2), pipeline);
                                }
                            }
                            else
                            {
                            }
                        }

                        stage++;
                        if (stage >= STAGES)
                            stage = 0;
                    }
                    pipeline.producer_commit();
                }
            }
            else
            {
                float state_reg[DSTATE];
                float A_reg[DSTATE];
                for (int i = 0; i < DSTATE; i++)
                {
                    state_reg[i] = 0.f;
                    A_reg[i] = toFloat(A[i * num_channels + channel]);
                }

                cuda::pipeline pipeline = cuda::make_pipeline(block, &pipeline_state, cuda::pipeline_role::consumer);
                int stage = 0;
                for (int si = 0; si < seq_loops; si++)
                {

                    pipeline.consumer_wait();

#pragma unroll
                    for (int token_id = si * SEQ_UNROLL; token_id < num_tokens && token_id < (si + 1) * SEQ_UNROLL; token_id++)
                    {

                        float dt_b = toFloat(sh_dt[stage][threadIdx.x]) + toFloat(sh_dt_bias[threadIdx.x]);
                        float dt_b_sp;
                        if (dt_softplus)
                        {
                            dt_b_sp = dt_b <= 20.f ? log1pf(__expf(dt_b)) : dt_b; // softplus
                        }
                        float my_x = toFloat(sh_x[stage][threadIdx.x]);
                        float Dx = my_x * (D ? toFloat(sh_D[threadIdx.x]) : 0.f);
                        float dtx = dt_b_sp * my_x;
                        float my_z = z ? toFloat(sh_z[stage][threadIdx.x]) : 0.f;

                        float out = Dx;

                        if (sizeof(input_t) == 4)
                        {
                            float4* B4 = (float4*)&sh_B[stage][0];
                            float4* C4 = (float4*)&sh_C[stage][0];
#pragma unroll
                            for (int i = 0; i < DSTATE / 4; i++)
                            {

                                float4 Bi4 = B4[i];
                                float4 Ci4 = C4[i];

                                float* Bi = (float*)&Bi4;
                                float* Ci = (float*)&Ci4;

#pragma unroll
                                for (int j = 0; j < 4; j++)
                                {
                                    float dtA = A_reg[i * 4 + j] * dt_b_sp;
                                    float dA = __expf(dtA);
                                    float sdA = state_reg[i * 4 + j] * dA;
                                    float dBx = Bi[j] * dtx;
                                    float newState = sdA + dBx;
                                    state_reg[i * 4 + j] = newState;
                                    out += newState * Ci[j];
                                }
                            }
                        }
                        else
                        {
                            float4* B8 = (float4*)&sh_B[stage][0];
                            float4* C8 = (float4*)&sh_C[stage][0];
#pragma unroll
                            for (int i = 0; i < DSTATE / 8; i++)
                            {
                                input_t* Bi = (input_t*)(&B8[i]);
                                input_t* Ci = (input_t*)(&C8[i]);
#pragma unroll
                                for (int j = 0; j < 8; j++)
                                {
                                    float dtA = A_reg[i * 8 + j] * dt_b_sp;
                                    float dA = __expf(dtA);
                                    float sdA = state_reg[i * 8 + j] * dA;
                                    float dBx = toFloat(Bi[j]) * dtx;
                                    float newState = sdA + dBx;
                                    state_reg[i * 8 + j] = newState;
                                    out += newState * toFloat(Ci[j]);
                                }
                            }
                        }

                        if (z)
                        {
                            float enz = __expf(0.f - my_z);
                            enz += 1.0;
                            float sig_z = 1.0 / enz;
                            float silu_z = my_z * sig_z;
                            out *= silu_z;
                        }
                        input_t* my_output = &output[input_matrix_row_id * num_channels + token_id * num_channels];
                        convertAndStore(&my_output[channel], out);

                        stage++;
                        if (stage >= STAGES)
                            stage = 0;
                    }
                    pipeline.consumer_release();
                }
                for (int i = 0; i < DSTATE; i++)
                {
                    input_t* my_state = &state[sample * num_channels * DSTATE];
                    int offset = i * num_channels + channel;
                    convertAndStore(&my_state[offset], state_reg[i]);
                }
            }
        }

        template <typename input_t, typename weight_t>
        void invokeSelectiveScan(SSMParamsBase& params, hipStream_t stream)
        {
            int samples = params.batch;
            int channels = params.dim;

            int const threads = 128;
            int const blocks = (channels + threads - 1) / threads;
            dim3 block(threads, 2);
            dim3 grid(blocks, samples);
            CHECK((channels % block.x) == 0);

            CHECK(params.is_variable_B);
            CHECK(params.is_variable_C);
            CHECK(params.dstate == 16);

            selective_scan_loop_kernel<input_t, weight_t> << <grid, block, 0, stream >> > (params);
        }

#define INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(input_t, weight_t)                                                        \
    template void invokeSelectiveScan<input_t, weight_t>(SSMParamsBase & params, hipStream_t stream);

        INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(float, float);
        INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
        INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE

        template <typename input_t, typename weight_t, int DSTATE = 16, int CHANNELS_PER_BLOCK = 128>
        __launch_bounds__(128, 2) __global__ void selective_scan_update_kernel(SSMParamsBase params)
        {

            input_t* output = reinterpret_cast<input_t*>(params.out_ptr);
            input_t* state = reinterpret_cast<input_t*>(params.x_ptr);
            input_t* x = reinterpret_cast<input_t*>(params.u_ptr);
            input_t* dt = reinterpret_cast<input_t*>(params.delta_ptr);
            weight_t* A = reinterpret_cast<weight_t*>(params.A_ptr);
            input_t* B = reinterpret_cast<input_t*>(params.B_ptr);
            input_t* C = reinterpret_cast<input_t*>(params.C_ptr);
            weight_t* D = reinterpret_cast<weight_t*>(params.D_ptr);
            input_t* z = reinterpret_cast<input_t*>(params.z_ptr);
            weight_t* dt_bias = reinterpret_cast<weight_t*>(params.delta_bias_ptr);
            bool dt_softplus = params.delta_softplus;
            int num_channels = params.dim;

            int const channel = blockIdx.x * blockDim.x + threadIdx.x;
            if (channel >= num_channels)
                return;
            int const sample = blockIdx.y;

            input_t* my_state = &state[sample * num_channels * DSTATE];
            input_t* my_output = &output[sample * num_channels];

            float rA[DSTATE];
            float rB[DSTATE];
            float rC[DSTATE];

            float rState[DSTATE];

#pragma unroll
            for (int i = 0; i < DSTATE; i++)
            {
                rA[i] = toFloat(A[i * num_channels + channel]);
                rB[i] = toFloat(B[sample * DSTATE + i]);
                rC[i] = toFloat(C[sample * DSTATE + i]);
                rState[i] = toFloat(my_state[i * num_channels + channel]);
            }

            float my_x, my_dt, my_z, my_dt_bias, my_D;
            my_x = toFloat(x[sample * num_channels + channel]);
            my_dt = toFloat(dt[sample * num_channels + channel]);
            my_z = z ? toFloat(z[sample * num_channels + channel]) : 0.f;
            my_dt_bias = dt_bias ? toFloat(dt_bias[channel]) : 0.f;
            my_D = D ? toFloat(D[channel]) : 0.f;

            float dt_b = my_dt + my_dt_bias;
            float dt_b_sp;
            if (dt_softplus)
            {
                dt_b_sp = dt_b <= 20.f ? logf(1.f + expf(dt_b)) : dt_b;
            }

            float out = 0.f;

#pragma unroll
            for (int i = 0; i < DSTATE; i++)
            {
                float dA = expf(rA[i] * dt_b_sp);
                float dB = rB[i] * dt_b_sp;
                float sdA = rState[i] * dA;
                float dBx = dB * my_x;
                float newState = sdA + dBx;
                convertAndStore(&my_state[i * num_channels + channel], newState);
                out += newState * rC[i];
            }

            if (D)
                out += my_D * my_x;
            if (z)
            {
                float sig_z = 1.0 / (1.0 + exp(0.f - my_z));
                float silu_z = my_z * sig_z;
                out *= silu_z;
            }

            convertAndStore(&my_output[channel], out);
        }

        template <typename input_t, typename weight_t>
        void invokeSelectiveScanUpdate(SSMParamsBase& params, hipStream_t stream)
        {
            int samples = params.batch;
            int channels = params.dim;

            int const threads = 128;
            int const blocks = (channels + threads - 1) / threads;
            dim3 block(threads, 1);
            dim3 grid(blocks, samples);

            CHECK(params.is_variable_B);
            CHECK(params.is_variable_C);
            CHECK(params.dstate == 16);
            selective_scan_update_kernel<input_t, weight_t> << <grid, block, 0, stream >> > (params);
        }

#define INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(input_t, weight_t)                                                 \
    template void invokeSelectiveScanUpdate<input_t, weight_t>(SSMParamsBase & params, hipStream_t stream)

        INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(float, float);
        INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
        INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE

    }
}