#include "hip/hip_runtime.h"
#include "../common/cudaUtils.h"
#include "banBadWords.h"

using namespace nexly::common;

namespace nexly
{
    namespace nlp
    {

        /// <summary>
        /// Kernel function to ban bad words from the logits.
        /// </summary>
        /// <typeparam name="T">Type of the logits.</typeparam>
        /// <param name="logits">The logits array.</param>
        /// <param name="output_ids_ptr">Pointer to the output ids.</param>
        /// <param name="parent_ids_ptr">Pointer to the parent ids.</param>
        /// <param name="batch_slots">Array containing batch slots.</param>
        /// <param name="beam_width">Beam width.</param>
        /// <param name="bad_words_ptrs">Array of pointers to bad words.</param>
        /// <param name="bad_words_lens">Array containing lengths of bad words.</param>
        /// <param name="vocab_size_padded">Padded vocabulary size.</param>
        /// <param name="sequence_lengths">Array containing lengths of sequences.</param>
        /// <param name="max_seq_len">Maximum sequence length.</param>
        template <typename T>
        __global__ void ban_bad_words(T* logits, int32_t const** output_ids_ptr, int32_t const** parent_ids_ptr,
            int32_t const* batch_slots, int32_t beam_width, int32_t const** bad_words_ptrs, int32_t const* bad_words_lens,
            int32_t vocab_size_padded, int32_t const* sequence_lengths, const int32_t max_seq_len)
        {
            int32_t const id = blockIdx.x * blockDim.x + threadIdx.x;
            int32_t const batch_idx = blockIdx.y / beam_width;
            int32_t const beam_idx = blockIdx.y % beam_width;
            auto const batch_slot = batch_slots != nullptr ? batch_slots[batch_idx] : batch_idx;
            auto const batch_beam_idx = batch_slot * beam_width + beam_idx;

            int32_t const* base_bad_words = bad_words_ptrs[batch_slot];
            auto const bad_words_len = bad_words_lens[batch_slot];
            int32_t const* base_bad_words_offsets = base_bad_words + bad_words_len;

            if (id >= bad_words_len || base_bad_words_offsets[id] < 0)
            {
                return;
            }

            auto const item_end = base_bad_words_offsets[id];
            auto const item_start = (id > 0) ? base_bad_words_offsets[id - 1] : 0;
            auto const item_size = item_end - item_start;

            bool should_ban = item_size == 1;
            int32_t const current_step{ sequence_lengths[batch_beam_idx] };

            if (item_size > 1 && current_step >= item_size - 1)
            {
                should_ban = true;
                int32_t parent_id = beam_idx;
                bool const gather_beam = beam_width > 1;

                for (int32_t token_idx = item_size - 2; token_idx >= 0; token_idx--)
                {
                    auto const previous_token
                        = output_ids_ptr[batch_slot][parent_id * max_seq_len + current_step - (item_size - 1) + token_idx];

                    if (previous_token != base_bad_words[item_start + token_idx])
                    {
                        should_ban = false;
                        break;
                    }
                    if (gather_beam)
                    {
                        parent_id = parent_ids_ptr == nullptr
                            ? 0
                            : parent_ids_ptr[batch_slot][parent_id * max_seq_len + current_step - (item_size - 1) + token_idx];

                        if (parent_id < 0 || parent_id >= beam_width)
                        {
                            should_ban = false;
                            break;
                        }
                    }
                }
            }

            if (should_ban)
            {
                auto banned_token = base_bad_words[item_end - 1];
                if (0 <= banned_token && banned_token < vocab_size_padded)
                {
                    logits[batch_idx * beam_width * vocab_size_padded + beam_idx * vocab_size_padded + banned_token]
                        = static_cast<T>(-INFINITY);
                }
            }
        }

        /// <summary>
        /// Invokes the function to ban bad words from the logits.
        /// </summary>
        /// <typeparam name="T">Type of the logits.</typeparam>
        /// <param name="logits">The logits array.</param>
        /// <param name="output_ids_ptr">Pointer to the output ids.</param>
        /// <param name="parent_ids_ptr">Pointer to the parent ids.</param>
        /// <param name="batch_slot">Batch slot.</param>
        /// <param name="batch_size">Batch size.</param>
        /// <param name="beam_width">Beam width.</param>
        /// <param name="bad_words">Bad words array.</param>
        /// <param name="bad_words_lens">Array containing lengths of bad words.</param>
        /// <param name="max_bad_words_len">Maximum length of bad words.</param>
        /// <param name="vocab_size_padded">Padded vocabulary size.</param>
        /// <param name="sequence_lengths">Array containing lengths of sequences.</param>
        /// <param name="max_seq_len">Maximum sequence length.</param>
        /// <param name="stream">CUDA stream.</param>
        template <typename T>
        void invokeBanBadWords(T* logits, int32_t const** output_ids_ptr, int32_t const** parent_ids_ptr,
            int32_t const* batch_slot, int32_t batch_size, int32_t beam_width, int32_t const** bad_words,
            int32_t const* bad_words_lens, int32_t max_bad_words_len, int32_t vocab_size_padded,
            int32_t const* sequence_lengths, int32_t max_seq_len, hipStream_t stream)
        {
            dim3 block, grid;
            constexpr int32_t max_blocks{ 256 };
            block.x = min(((max_bad_words_len + 32 - 1) / 32) * 32, max_blocks);
            grid.x = (max_bad_words_len + block.x - 1) / block.x;
            grid.y = batch_size * beam_width;

            ban_bad_words << <grid, block, 0, stream >> > (logits, output_ids_ptr, parent_ids_ptr, batch_slot, beam_width, bad_words,
                bad_words_lens, vocab_size_padded, sequence_lengths, max_seq_len);
            sync_check_cuda_error();
        }

        template void invokeBanBadWords(half* logits, int32_t const** output_ids_ptr, int32_t const** parent_ids_ptr,
            int32_t const* batch_slot, int32_t batch_size, int32_t beam_width, int32_t const** bad_words,
            int32_t const* bad_words_lens, int32_t max_bad_words_len, int32_t vocab_size_padded,
            int32_t const* sequence_lengths, int32_t max_seq_len, hipStream_t stream);
#ifdef ENABLE_BF16
        template void invokeBanBadWords(__hip_bfloat16* logits, int32_t const** output_ids_ptr, int32_t const** parent_ids_ptr,
            int32_t const* batch_slot, int32_t batch_size, int32_t beam_width, int32_t const** bad_words,
            int32_t const* bad_words_lens, int32_t max_bad_words_len, int32_t vocab_size_padded,
            int32_t const* sequence_lengths, int32_t max_seq_len, hipStream_t stream);
#endif
        template void invokeBanBadWords(float* logits, int32_t const** output_ids_ptr, int32_t const** parent_ids_ptr,
            int32_t const* batch_slot, int32_t batch_size, int32_t beam_width, int32_t const** bad_words,
            int32_t const* bad_words_lens, int32_t max_bad_words_len, int32_t vocab_size_padded,
            int32_t const* sequence_lengths, int32_t max_seq_len, hipStream_t stream);

    }
}
